#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "psrdada_cpp/cuda_utils.hpp"
#include "skyweaver/BeamformerPipeline.cuh"

#include <cstdlib>
#include <exception>
#include <stdexcept>
#include <string>


namespace skyweaver
{

template <typename CBHandler,
          typename IBHandler,
          typename StatsHandler,
          typename BeamformerTraits>
BeamformerPipeline<CBHandler, IBHandler, StatsHandler, BeamformerTraits>::
    BeamformerPipeline(PipelineConfig const& config,
                       CBHandler& cb_handler,
                       IBHandler& ib_handler,
                       StatsHandler& stats_handler)
    : _config(config), _nbeamsets(0), _cb_handler(cb_handler),
      _ib_handler(ib_handler), _stats_handler(stats_handler),
      _unix_timestamp(0.0), _call_count(0)
{
    BOOST_LOG_TRIVIAL(debug) << "Constructing beanmformer pipeline";
    std::size_t nsamples = _config.gulp_length_samps();
    BOOST_LOG_TRIVIAL(debug)
        << "Expected gulp size: " << nsamples << " (samples)";
    if(nsamples % _config.nsamples_per_heap() != 0) {
        throw std::runtime_error("Gulp size is not a multiple of "
                                 "the number of samples per heap");
    }
    std::size_t nheap_groups = nsamples / _config.nsamples_per_heap();
    std::size_t input_taftp_size =
        nheap_groups * nsamples / _config.nsamples_per_heap();
    _taftp_from_host.resize(input_taftp_size, {0, 0});
    // Calculate the timestamp step per block
    _sample_clock_tick_per_block = 2 * _config.total_nchans() * nsamples;
    BOOST_LOG_TRIVIAL(debug)
        << "Sample clock tick per block: " << _sample_clock_tick_per_block;
    CUDA_ERROR_CHECK(hipStreamCreate(&_h2d_copy_stream));
    CUDA_ERROR_CHECK(hipStreamCreate(&_processing_stream));
    CUDA_ERROR_CHECK(hipStreamCreate(&_d2h_copy_stream));

    float f_low = _config.centre_frequency() - _config.bandwidth()/2.0f;
    float f_high =  _config.centre_frequency() + _config.bandwidth()/2.0f;
    float tsamp = _config.nchans() / _config.bandwidth();
    auto it = std::max_element(_config.coherent_dms().begin(), _config.coherent_dms().end());
    float max_dm = *it;
    float max_dm_delay = CoherentDedisperser::get_dm_delay(f_low, f_high, max_dm);
    CoherentDedisperser::createConfig(
        _dedisperser_config,  _config.gulp_length_samps(), max_dm_delay, 
        _config.nchans(), _config.npol(), _config.nantennas(), tsamp, 
        f_low, _config.bandwidth(), _config.coherent_dms());

    BOOST_LOG_TRIVIAL(debug) << "Constructing delay and weights managers";
    _delay_manager.reset(new DelayManager(_config, _h2d_copy_stream));
    _weights_manager.reset(new WeightsManager(_config, _processing_stream));
    _stats_manager.reset(new StatisticsCalculator(_config, _processing_stream));
    _transposer.reset(new Transposer(_config));
    _coherent_beamformer.reset(new CoherentBeamformer(_config));
    _coherent_dedisperser.reset(
        new CoherentDedisperser(_dedisperser_config));
    _incoherent_beamformer.reset(new IncoherentBeamformer(_config));
    _dispenser.reset(new BufferedDispenser(_config, _processing_stream));
    _nbeamsets = _delay_manager->nbeamsets();
    BOOST_LOG_TRIVIAL(debug)
        << "Delay model contains " << _nbeamsets << " beamsets";
}

template <typename CBHandler,
          typename IBHandler,
          typename StatsHandler,
          typename BeamformerTraits>
BeamformerPipeline<CBHandler, IBHandler, StatsHandler, BeamformerTraits>::
    ~BeamformerPipeline()
{
    CUDA_ERROR_CHECK(hipStreamDestroy(_h2d_copy_stream));
    CUDA_ERROR_CHECK(hipStreamDestroy(_processing_stream));
    CUDA_ERROR_CHECK(hipStreamDestroy(_d2h_copy_stream));
}

template <typename CBHandler,
          typename IBHandler,
          typename StatsHandler,
          typename BeamformerTraits>
void BeamformerPipeline<CBHandler, IBHandler, StatsHandler, BeamformerTraits>::
    init(ObservationHeader const& header)
{
    BOOST_LOG_TRIVIAL(debug) << "Initialising beamformer pipeline";
    _header = header;
    _cb_handler.init(_header);
    _ib_handler.init(_header);
    _stats_handler.init(_header);
    _taftp_from_host.resize(_config.gulp_length_samps() * header.nantennas *
                                _config.nchans() * _config.npol(),
                            {0, 0});
    BOOST_LOG_TRIVIAL(debug) << "Resized TAFTP input vector to "
                             << _taftp_from_host.size() << " elements";
}

template <typename CBHandler,
          typename IBHandler,
          typename StatsHandler,
          typename BeamformerTraits>
void BeamformerPipeline<CBHandler, IBHandler, StatsHandler, BeamformerTraits>::
    process()
{
    BOOST_LOG_TRIVIAL(debug) << "Executing beamforming pipeline";

    // Need to add the unix timestmap to the delay manager here
    // to fetch valid delays for this epoch.
    BOOST_LOG_TRIVIAL(debug) << "Checking for delay updates";
    
    _timer.start("fetch delays");
    auto const& delays = _delay_manager->delays(_unix_timestamp);
    _timer.stop("fetch delays");

    // Stays the same
    BOOST_LOG_TRIVIAL(debug)
        << "Calculating weights at unix time: " << _unix_timestamp;
    
    _timer.start("calculate weights");
    auto const& weights = _weights_manager->weights(delays,
                                                    _unix_timestamp,
                                                    _delay_manager->epoch());
    _timer.stop("calculate weights");


    BOOST_LOG_TRIVIAL(debug)
        << "Transposing input data from TAFTP to FTPA order";
    _timer.start("transpose TAFTP to FTPA");
    _transposer->transpose(_taftp_from_host,
                           _ftpa_post_transpose,
                           _header.nantennas,
                           _processing_stream);
    _timer.stop("transpose TAFTP to FTPA");                        
    _ftpa_dedispersed.resize(_ftpa_post_transpose.size());
    // Stays the same
    BOOST_LOG_TRIVIAL(debug) << "Checking if channel statistics update request";
    _timer.start("calculate statistics");
    _stats_manager->calculate_statistics(_ftpa_post_transpose);
    _timer.stop("calculate statistics");
    if (_call_count == 0)
    {   
        _timer.start("update scalings");
        _stats_manager->update_scalings(_delay_manager->beamset_weights(),
                                        _delay_manager->nbeamsets());
        _timer.stop("update scalings");
    }
    
    BOOST_LOG_TRIVIAL(debug)
        << "FTPA post transpose size: " << _ftpa_post_transpose.size();

    _timer.start("dispenser hoarding");
    _dispenser->hoard(_ftpa_post_transpose);
    _timer.stop("dispenser hoarding");

    for(unsigned int dm_idx = 0; dm_idx < _config.coherent_dms().size();
        ++dm_idx) {
        
        _timer.start("coherent dedispersion");
        for(unsigned int freq_idx = 0; freq_idx < _config.nchans();
            ++freq_idx) {
            BOOST_LOG_TRIVIAL(debug) << "{{{[[[<<< DM Idx: " << dm_idx
                                     << " F Idx: " << freq_idx << " >>>]]]}}}";
            BOOST_LOG_TRIVIAL(debug) << "Dispensing some voltages";
            auto const& tpa_voltages = _dispenser->dispense(freq_idx);
            BOOST_LOG_TRIVIAL(debug) << "Attempting to segfault";

            _coherent_dedisperser->dedisperse(
                tpa_voltages,
                _ftpa_dedispersed,
                freq_idx ,
                dm_idx);
        }
        _timer.stop("coherent dedispersion");

        BOOST_LOG_TRIVIAL(debug) << "_ftpa_dedispersed.size() = " << _ftpa_dedispersed.size();
        BOOST_LOG_TRIVIAL(debug) << "_stats_manager->ib_scaling() = " << _stats_manager->ib_scaling().size();
        BOOST_LOG_TRIVIAL(debug) << "_stats_manager->ib_offsets() = " << _stats_manager->ib_offsets().size();
        BOOST_LOG_TRIVIAL(debug) << "_delay_manager->beamset_weights() = " << _delay_manager->beamset_weights().size();

        _timer.start("incoherent beamforming");
        _incoherent_beamformer->beamform(_ftpa_dedispersed,
                                         _tf_ib_raw,
                                         _tf_ib,
                                         _stats_manager->ib_scaling(),
                                         _stats_manager->ib_offsets(),
                                         _delay_manager->beamset_weights(),
                                         _nbeamsets,
                                         _processing_stream);
        _timer.stop("incoherent beamforming");
        _timer.start("coherent beamforming");
        _coherent_beamformer->beamform(_ftpa_dedispersed,
                                       weights,
                                       _stats_manager->cb_scaling(),
                                       _stats_manager->cb_offsets(),
                                       _delay_manager->beamset_mapping(),
                                       _tf_ib_raw,
                                       _btf_cbs,
                                       _nbeamsets,
                                       _processing_stream);
        _timer.stop("coherent beamforming");
        _timer.start("coherent beam handler");
        _cb_handler(_btf_cbs, dm_idx);
        _timer.stop("coherent beam handler");
        _timer.start("incoherent beam handler");
        _ib_handler(_tf_ib, dm_idx);
        _timer.stop("incoherent beam handler");
    }
    _timer.start("statistics handler");
    _stats_handler(_stats_manager->statistics());
    _timer.stop("statistics handler");
    _timer.show_all_timings();
}

template <typename CBHandler,
          typename IBHandler,
          typename StatsHandler,
          typename BeamformerTraits>
bool BeamformerPipeline<CBHandler, IBHandler, StatsHandler, BeamformerTraits>::
operator()(HostVoltageVectorType const& taftp_on_host)
{
    BOOST_LOG_TRIVIAL(debug) << "Pipeline operator() called";
    BOOST_LOG_TRIVIAL(debug) << "taftp_on_host size: " << taftp_on_host.size();

    if(taftp_on_host.size() != _taftp_from_host.size()) {
        throw std::runtime_error(
            std::string("Unexpected buffer size, expected ") +
            std::to_string(taftp_on_host.size()) + " but got " +
            std::to_string(_taftp_from_host.size()));
    }

    CUDA_ERROR_CHECK(hipMemcpyAsync(
        static_cast<void*>(thrust::raw_pointer_cast(_taftp_from_host.data())),
        static_cast<void const*>(
            thrust::raw_pointer_cast(taftp_on_host.data())),
        taftp_on_host.size() * sizeof(char2),
        hipMemcpyHostToDevice,
        _h2d_copy_stream));
    CUDA_ERROR_CHECK(hipStreamSynchronize(_h2d_copy_stream));

    // Calculate the unix timestamp for the block that is about to be
    // processed
    _unix_timestamp =
        _header.utc_start +
        static_cast<long double>(_call_count * _sample_clock_tick_per_block) /
            _header.sample_clock;
    process();
    CUDA_ERROR_CHECK(hipStreamSynchronize(_processing_stream));
    CUDA_ERROR_CHECK(hipStreamSynchronize(_d2h_copy_stream));
    ++_call_count;
    return false;
}
} // namespace skyweaver