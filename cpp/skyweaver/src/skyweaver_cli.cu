#include "hip/hip_runtime.h"
#include "boost/program_options.hpp"
#include "errno.h"
#include "psrdada_cpp/cli_utils.hpp"
#include "skyweaver/BeamformerPipeline.cuh"
#include "skyweaver/IncoherentDedispersionPipeline.cuh"
#include "skyweaver/MultiFileReader.cuh"
#include "skyweaver/MultiFileWriter.cuh"
#include "skyweaver/PipelineConfig.hpp"
#include "skyweaver/DescribedVector.hpp"
#include "thrust/host_vector.h"
#include "thrust/device_vector.h"

#include <algorithm>
#include <cerrno>
#include <fstream>
#include <ios>
#include <iostream>
#include <sstream>
#include <string>
#include <sys/types.h>
#include <vector>

#define BOOST_LOG_DYN_LINK 1

namespace
{
const size_t ERROR_IN_COMMAND_LINE     = 1;
const size_t SUCCESS                   = 0;
const size_t ERROR_UNHANDLED_EXCEPTION = 2;

class NullHandler
{
  public:
    template <typename... Args>
    void init(Args... args) {};

    template <typename... Args>
    bool operator()(Args... args)
    {
        return false;
    };
};
} // namespace

// This patching of the << operator is required to allow
// for float vector arguments to boost program options
namespace std
{
std::ostream& operator<<(std::ostream& os, const std::vector<float>& vec)
{
    for(auto item: vec) { os << item << " "; }
    return os;
}
} // namespace std

template <class Pipeline>
void run_pipeline(Pipeline& pipeline, skyweaver::PipelineConfig& config){

    skyweaver::MultiFileReader file_reader(config);
    auto const& header = file_reader.get_header();
    validate_header(header, config);
    update_config(config, header);
    std::size_t input_elements = header.nantennas * config.nchans() *
                                 config.npol() * config.gulp_length_samps();
    
    typename Pipeline::HostVoltageVectorType taftp_input_voltage({
        config.gulp_length_samps()/config.nsamples_per_heap(), // T
        header.nantennas, // A
        config.nchans(), // F
        config.nsamples_per_heap(), // T
        config.npol()
        });
    taftp_input_voltage.frequencies(config.channel_frequencies());
    taftp_input_voltage.dms({0.0f});
    taftp_input_voltage.tsamp(header.obs_nchans / header.obs_bandwidth);

    std::size_t input_bytes = taftp_input_voltage.size() * sizeof(typename decltype(taftp_input_voltage)::value_type);
    pipeline.init(header);

    BOOST_LOG_TRIVIAL(info)
        << "Total input size (bytes): " << file_reader.get_total_size();
    // TODO: Add a parameter to PipelineConfig for start sample? time?
    // TODO: Add a parameter to PipelineConfig for nsamples? duration?
    while(!file_reader.eof()) {
        std::streamsize nbytes_read =
            file_reader.read(reinterpret_cast<char*>(thrust::raw_pointer_cast(
                                 taftp_input_voltage.data())),
                             input_bytes);
        pipeline(taftp_input_voltage);
    }
}

template <typename BfTraits, bool enable_incoherent_dedispersion>
void setup_pipeline(skyweaver::PipelineConfig& config)
{
    using OutputType = typename BfTraits::QuantisedPowerType;
    //NullHandler ib_handler;
    NullHandler stats_handler;
    skyweaver::MultiFileWriter<skyweaver::BTFPowersH<OutputType>> ib_handler(config, "ib");
    if constexpr (enable_incoherent_dedispersion)
    {   
        skyweaver::MultiFileWriter<skyweaver::TDBPowersH<OutputType>> cb_file_writer(config, "cb");
        skyweaver::IncoherentDedispersionPipeline<OutputType, OutputType, decltype(cb_file_writer)> dispersion_pipeline(config, cb_file_writer);
        skyweaver::BeamformerPipeline<decltype(dispersion_pipeline),
                                      decltype(ib_handler),
                                      decltype(stats_handler),
                                      BfTraits>
            pipeline(config, dispersion_pipeline, ib_handler, stats_handler);
        run_pipeline(pipeline, config);
    } else {
        skyweaver::MultiFileWriter<skyweaver::TFBPowersD<OutputType>> cb_file_writer(config, "cb");
        skyweaver::BeamformerPipeline<decltype(cb_file_writer),
                                      decltype(ib_handler),
                                      decltype(stats_handler),
                                      BfTraits>
            pipeline(config, cb_file_writer, ib_handler, stats_handler);
        run_pipeline(pipeline, config);
    }
}

int main(int argc, char** argv)
{
    try {
        skyweaver::PipelineConfig config;

        /**
         * Define and parse the program options
         */
        namespace po = boost::program_options;

        // Generic options group here to contain the configuration file name
        // The config file will be parsed AFTER the the command line options
        // this gives command line options precedence over config file options.
        // Options here are only settable via the command line.
        po::options_description generic("Generic options");
        generic.add_options()("cfg,c",
                              po::value<std::string>()->default_value(""),
                              "Skyweaver configuration file");

        // Main option group that contains parameters settable via both the
        // command line and the config file
        po::options_description main_options("Main options");
        main_options.add_options()

            // Help menu
            ("help,h", "Display help messages")

            // Input file containing list of DADA files to process
            ("input-file",
             po::value<std::string>()->required()->notifier(
                 [&config](std::string key) {
                     config.read_input_file_list(key);
                 }),
             "File containing list of DADA files to process")

            // Input file for delay solutions
            // This can contain any number of beams but only beams
            // up to the maximum configured for generation will be
            // produced. Antenna ordering in the file must match
            // the antenna order of the input data.
            ("delay-file",
             po::value<std::string>()->required()->notifier(
                 [&config](std::string key) { config.delay_file(key); }),
             "File containing delay solutions")

            // Output file for block statistics
            ("stats-file",
             po::value<std::string>()
                 ->default_value(config.statistics_file())
                 ->notifier([&config](std::string key) {
                     config.statistics_file(key);
                 }),
             "Output file for block statistics")

            // Output directory where all results will be written
            ("output-dir",
             po::value<std::string>()
                 ->default_value(config.output_dir())
                 ->notifier(
                     [&config](std::string key) { config.output_dir(key); }),
             "The output directory for all results")

            // Output file for block statistics
            ("output-level",
             po::value<float>()
                 ->default_value(config.output_level())
                 ->notifier([&config](float key) { config.output_level(key); }),
             "The desired standard deviation for output data")

            /**
             * Dispersion measures for coherent dedispersion
             * Can be specified on the command line with:
             *
             * --coherent-dm 1 2 3
             * or
             * --coherent-dm 1 --coherent-dm 2 --coherent-dm 3
             *
             * In the configuration file it can only be specified with:
             *
             * coherent-dm=1
             * coherent-dm=2
             * coherent-dm=3
             */
            ("coherent-dm",
             po::value<std::vector<float>>()
                 ->multitoken()
                 ->default_value(config.coherent_dms())
                 ->notifier([&config](std::vector<float> const& dms) {
                     config.coherent_dms(dms);
                 }),
             "The dispersion measures to coherently dedisperse to")

            ("enable-incoherent-dedispersion",
             po::value<bool>()
                 ->default_value(true)
                 ->notifier([&config](bool const& enable) {
                     config.enable_incoherent_dedispersion(enable);
                 }),
             "Turn on/off incoherent dedispersion after beamforming")

            // Number of samples to read in each gulp
            ("gulp-size",
             po::value<std::size_t>()
                 ->default_value(config.gulp_length_samps())
                 ->notifier([&config](std::size_t const& gulp_size) {
                     // Round off to next multiple of 256
                     if(gulp_size % config.nsamples_per_heap() != 0) {
                         BOOST_LOG_TRIVIAL(debug)
                             << "Rounding up gulp-size to next multiple of 256";
                         config.gulp_length_samps(
                             (gulp_size / config.nsamples_per_heap()) *
                             config.nsamples_per_heap());
                     } else {
                         config.gulp_length_samps(gulp_size);
                     }
                 }),
             "The number of samples to read in each gulp ")

            // Stokes mode I, Q, U, V or IQUV
            ("stokes-mode",
             po::value<std::string>()->default_value("I")->notifier(
                 [&config](std::string stokes) {
                     for(auto& c: stokes) c = (char)toupper(c);
                     config.stokes_mode(stokes);
                 }),
             "The Stokes mode to use, can be either I, Q, U, V or IQUV")

            // Logging options
            ("log-level",
             po::value<std::string>()->default_value("info")->notifier(
                 [](std::string level) { psrdada_cpp::set_log_level(level); }),
             "The logging level to use (debug, info, warning, error)");

        // set options allowed on command line
        po::options_description cmdline_options;
        cmdline_options.add(generic).add(main_options);

        // set options allowed in config file
        po::options_description config_file_options;
        config_file_options.add(main_options);

        po::variables_map variable_map;
        try {
            po::store(po::command_line_parser(argc, argv)
                          .options(cmdline_options)
                          .run(),
                      variable_map);
            if(variable_map.count("help")) {
                std::cout << "skyweavercpp -- C++/CUDA beamformer pipeline for "
                             "COMPACT-ERC"
                          << std::endl
                          << cmdline_options << std::endl;
                return SUCCESS;
            }
        } catch(po::error& e) {
            std::cerr << "ERROR: " << e.what() << std::endl << std::endl;
            return ERROR_IN_COMMAND_LINE;
        }

        auto config_file = variable_map.at("cfg").as<std::string>();

        if(config_file != "") {
            std::ifstream config_fs(config_file.c_str());
            if(!config_fs.is_open()) {
                std::cerr << "Unable to open configuration file: "
                          << config_file << " (" << std::strerror(errno)
                          << ")\n";
                return ERROR_UNHANDLED_EXCEPTION;
            } else {
                po::store(po::parse_config_file(config_fs, config_file_options),
                          variable_map);
            }
        }
        po::notify(variable_map);

        /**
         * All the application code goes here
         */
        BOOST_LOG_TRIVIAL(info)
            << "Initialising the skyweaver beamforming pipeline";
        if(config_file != "") {
            BOOST_LOG_TRIVIAL(info) << "Configuration file: " << config_file;
        }
        BOOST_LOG_TRIVIAL(info)
            << "Input file count: " << config.input_files().size();
        BOOST_LOG_TRIVIAL(info) << "Delay file: " << config.delay_file();
        BOOST_LOG_TRIVIAL(info) << "Stats file: " << config.statistics_file();
        BOOST_LOG_TRIVIAL(info) << "Output dir: " << config.output_dir();
        BOOST_LOG_TRIVIAL(info) << "Output level: " << config.output_level();
        BOOST_LOG_TRIVIAL(info) << "Coherent DMs: " << config.coherent_dms();
        BOOST_LOG_TRIVIAL(info) << "Gulp size: " << config.gulp_length_samps();

        if (config.enable_incoherent_dedispersion())
        {
            if(config.stokes_mode() == "I") {
                setup_pipeline<skyweaver::SingleStokesBeamformerTraits<
                    skyweaver::StokesParameter::I>, true>(config);
            } else if(config.stokes_mode() == "Q") {
                setup_pipeline<skyweaver::SingleStokesBeamformerTraits<
                    skyweaver::StokesParameter::Q>, true>(config);
            } else if(config.stokes_mode() == "U") {
                setup_pipeline<skyweaver::SingleStokesBeamformerTraits<
                    skyweaver::StokesParameter::U>, true>(config);
            } else if(config.stokes_mode() == "V") {
                setup_pipeline<skyweaver::SingleStokesBeamformerTraits<
                    skyweaver::StokesParameter::V>, true>(config);
            } else if(config.stokes_mode() == "IQUV") {
                setup_pipeline<skyweaver::FullStokesBeamformerTraits, true>(config);
            } else {
                throw std::runtime_error("Invalid Stokes mode passed, must be one "
                                        "of I, Q, U, V or IQUV");
            }
        } else {
            if(config.stokes_mode() == "I") {
                setup_pipeline<skyweaver::SingleStokesBeamformerTraits<
                    skyweaver::StokesParameter::I>, false>(config);
            } else if(config.stokes_mode() == "Q") {
                setup_pipeline<skyweaver::SingleStokesBeamformerTraits<
                    skyweaver::StokesParameter::Q>, false>(config);
            } else if(config.stokes_mode() == "U") {
                setup_pipeline<skyweaver::SingleStokesBeamformerTraits<
                    skyweaver::StokesParameter::U>, false>(config);
            } else if(config.stokes_mode() == "V") {
                setup_pipeline<skyweaver::SingleStokesBeamformerTraits<
                    skyweaver::StokesParameter::V>, false>(config);
            } else if(config.stokes_mode() == "IQUV") {
                setup_pipeline<skyweaver::FullStokesBeamformerTraits, false>(config);
            } else {
                throw std::runtime_error("Invalid Stokes mode passed, must be one "
                                        "of I, Q, U, V or IQUV");
            }
        }
    } catch(std::exception& e) {
        std::cerr << "Unhandled Exception reached the top of main: " << e.what()
                  << ", application will now exit" << std::endl;
        return ERROR_UNHANDLED_EXCEPTION;
    }
    return SUCCESS;
}