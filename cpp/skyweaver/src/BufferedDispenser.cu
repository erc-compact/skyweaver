#include "skyweaver/BufferedDispenser.cuh"
using namespace skyweaver;
BufferedDispenser::BufferedDispenser(PipelineConfig const& config,
                                    CoherentDedisperserConfig const& dedisp_config,
                                     hipStream_t stream)
    : _config(config), _stream(stream)
{
    this->_block_length_tpa =
        _config.nantennas() * _config.npol() * _config.gulp_length_samps();
    this->_max_delay_tpa =
        dedisp_config.overlap_samps * _config.nantennas() * _config.npol();

    BOOST_LOG_TRIVIAL(debug) << "BD -> Block length TPA: " << _block_length_tpa;
    BOOST_LOG_TRIVIAL(debug) << "BD -> Max delay TPA: " << _max_delay_tpa;

    // this->_d_prev_ftpa_voltages.resize(_nchans * _max_delay_tpa);

    _d_channeled_tpa_voltages.resize(_config.nchans());
    _d_prev_channeled_tpa_voltages.resize(_config.nchans());
    _first_hoard.resize(_config.nchans(), true);

    for(std::size_t i = 0; i < _config.nchans(); i++) {
        _d_channeled_tpa_voltages[i].resize(
            { _config.gulp_length_samps() +  dedisp_config.overlap_samps,
             _config.npol(),
             _config.nantennas()});
        _d_prev_channeled_tpa_voltages[i].resize(
            { dedisp_config.overlap_samps,
             _config.npol(),
             _config.nantennas()});
    }

    BOOST_LOG_TRIVIAL(debug) << "overlap samples: " << dedisp_config.overlap_samps;

   
}

void BufferedDispenser::hoard(FTPAVoltagesTypeD const& new_ftpa_voltages_in)
{
    auto const& freqs = new_ftpa_voltages_in.frequencies();
    for(std::size_t i = 0; i < _config.nchans(); i++) {
        _d_channeled_tpa_voltages[i].frequencies(freqs[i]);
    }

    typename FTPAVoltagesTypeD::value_type zeros{};
    for(std::size_t i = 0; i < _config.nchans(); i++) {
        if(_first_hoard[i]) { // if first time set overlaps as zeros
            BOOST_LOG_TRIVIAL(debug)
                << "BD -> Filling TPA voltages " << i
                << " with zeros up to length " << _max_delay_tpa;


            thrust::fill(_d_channeled_tpa_voltages[i].begin(),
                         _d_channeled_tpa_voltages[i].begin() + _max_delay_tpa,
                         zeros);
            _first_hoard[i] = false;

        } else { // first add corresponding overlap to output
            BOOST_LOG_TRIVIAL(debug) << "BD -> Copying previous voltages of size: " << _d_prev_channeled_tpa_voltages[i].size();
            thrust::copy(_d_prev_channeled_tpa_voltages[i].begin(),
                         _d_prev_channeled_tpa_voltages[i].end(),
                         _d_channeled_tpa_voltages[i].begin());

        }
        // then add the input data
        BOOST_LOG_TRIVIAL(debug) << "BD -> Copying new voltages of size: " << _block_length_tpa << " starting at " << i * _block_length_tpa ;
        thrust::copy(new_ftpa_voltages_in.begin() + i * _block_length_tpa,
                     new_ftpa_voltages_in.begin() + (i + 1) * _block_length_tpa,
                     _d_channeled_tpa_voltages[i].begin() + _max_delay_tpa);

        // update the overlap for the next hoard
        BOOST_LOG_TRIVIAL(debug) << "BD -> Updating overlap to the data between " << (i + 1) * _block_length_tpa - _max_delay_tpa << " and " << (i + 1) * _block_length_tpa;
        thrust::copy(new_ftpa_voltages_in.begin() +
                         (i + 1) * _block_length_tpa - _max_delay_tpa,
                     new_ftpa_voltages_in.begin() + (i + 1) * _block_length_tpa,
                     _d_prev_channeled_tpa_voltages[i].begin());
    }
}

typename BufferedDispenser::TPAVoltagesTypeD const&
BufferedDispenser::dispense(std::size_t chan_idx) const
{ // implements overlapped buffering of data
    return _d_channeled_tpa_voltages[chan_idx];
}
