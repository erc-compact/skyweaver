#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hipfft/hipfft.h"
#include "skyweaver/CoherentDedisperser.cuh"
#include "skyweaver/dedispersion_utils.cuh"

#include <cmath>
#include <hip/hip_runtime.h>
#include <iomanip>
#include <iostream>
#include <thrust/complex.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sequence.h>
#include <thrust/transform.h>
#include <vector>
namespace skyweaver
{

void create_coherent_dedisperser_config(CoherentDedisperserConfig& config,
                                        PipelineConfig const& pipeline_config)
{
    // the centre frequency and bandwidth are for the bridge. This is taken from Observation Header (not from the user)
    float f_low =
        pipeline_config.centre_frequency() - pipeline_config.bandwidth() / 2.0f;

    float f_high = f_low + pipeline_config.bandwidth()/pipeline_config.nchans();
    
        // pipeline_config.centre_frequency() + pipeline_config.bandwidth() / 2.0f;
    float tsamp  = pipeline_config.nchans() / pipeline_config.bandwidth();

    

    if(pipeline_config.coherent_dms().empty()) {
        throw std::runtime_error("No coherent DMs specified");
    }

    auto it      = std::max_element(pipeline_config.coherent_dms().begin(),
                               pipeline_config.coherent_dms().end());
    float max_dm = *it;
    BOOST_LOG_TRIVIAL(debug) << "Constructing coherent dedisperser plan";
    std::size_t max_dm_delay_samps = DMSampleDelay(max_dm, f_low, tsamp)(f_high);

    if(max_dm_delay_samps > 2 * pipeline_config.gulp_length_samps()) {
        throw std::runtime_error(
            "Gulp length must be at least 2 times the maximum DM delay");
    }

    if(max_dm_delay_samps %2 !=0) {
        max_dm_delay_samps++;
    }

    create_coherent_dedisperser_config(config,
                                       pipeline_config.gulp_length_samps(),
                                       max_dm_delay_samps,
                                       pipeline_config.nchans(),
                                       pipeline_config.npol(),
                                       pipeline_config.nantennas(),
                                       tsamp,
                                       f_low,
                                       pipeline_config.bandwidth(),
                                       pipeline_config.coherent_dms());
}
/*
 * @brief      Create a new CoherentDedisperser object, mostly used only for
 * testing
 *
 * @param      config  The config reference
 */
void create_coherent_dedisperser_config(CoherentDedisperserConfig& config,
                                        std::size_t gulp_samps,
                                        std::size_t overlap_samps,
                                        std::size_t num_coarse_chans,
                                        std::size_t npols,
                                        std::size_t nantennas,
                                        double tsamp,
                                        double low_freq,
                                        double bw,
                                        std::vector<float> dms)
{
    config.gulp_samps       = gulp_samps;
    config.overlap_samps    = overlap_samps;
    config.num_coarse_chans = num_coarse_chans;
    config.npols            = npols;
    config.nantennas        = nantennas;
    config.tsamp            = tsamp;
    config.low_freq       = low_freq;
    config.bw             = bw;
    config.high_freq      = low_freq + bw;
    config.coarse_chan_bw = bw / num_coarse_chans;
    config.filter_delay = tsamp * overlap_samps / 2.0;
    BOOST_LOG_TRIVIAL(warning) << "tsamp in create_coherent_dedisperser_config: " << config.tsamp;
    BOOST_LOG_TRIVIAL(warning) << "overlap_samps in create_coherent_dedisperser_config: " << config.overlap_samps;
    BOOST_LOG_TRIVIAL(warning) << "Filter delay: " << config.filter_delay;


    /* Precompute DM constants */
    config._h_dms = dms;
    config._d_dms = config._h_dms;
    config._d_dm_prefactor.resize(dms.size());
    config._d_ism_responses.resize(dms.size());
    for(int i = 0; i < dms.size(); i++) {
        config._d_ism_responses[i].resize(num_coarse_chans * gulp_samps);
    }

    thrust::transform(config._d_dms.begin(),
                      config._d_dms.end(),
                      config._d_dm_prefactor.begin(),
                      DMPrefactor());

    config.fine_chan_bw = config.coarse_chan_bw / config.gulp_samps;

    for(int idx = 0; idx < config._d_dms.size(); idx++) {
        get_dm_responses(config,
                         config._d_dm_prefactor[idx],
                         config._d_ism_responses[idx]);
    }

    // data is FTPA order, we will loop over F, so we are left with TPA order.
    // Let's fuse PA to X, so TX order.
    //  We stride and batch over X and transform T
    std::size_t X  = config.npols * config.nantennas;
    std::size_t fft_size  = config.gulp_samps + config.overlap_samps;
    int n[1]       = {static_cast<int>(fft_size)}; // FFT size
    int inembed[1] = {static_cast<int>(fft_size)};
    int onembed[1] = {static_cast<int>(fft_size)};
    int istride    = X;
    int ostride    = X;
    int idist      = 1;
    int odist      = 1;
    int batch      = X;

    if(hipfftPlanMany(&config._fft_plan,
                     1,
                     n,
                     inembed,
                     istride,
                     idist,
                     onembed,
                     ostride,
                     odist,
                     HIPFFT_C2C,
                     batch) != HIPFFT_SUCCESS) {
        std::runtime_error("CUFFT error: Plan creation failed");
    }

    BOOST_LOG_TRIVIAL(debug) << "FFT plan created";
}


namespace
{
#define NCHANS_PER_BLOCK 128
} // namespace

void CoherentDedisperser::dedisperse(
    TPAVoltagesD<char2> const& d_tpa_voltages_in,
    FTPAVoltagesD<char2>& d_ftpa_voltages_out,
    unsigned int freq_idx,
    unsigned int dm_idx)
{
    BOOST_LOG_NAMED_SCOPE("CoherentDedisperser::dedisperse");
    _d_fpa_spectra.resize(d_tpa_voltages_in.size(), {0.0f, 0.0f});
    _d_tpa_voltages_in_cufft.resize(d_tpa_voltages_in.size(), {0.0f, 0.0f});
    _d_tpa_voltages_dedispersed.resize(d_tpa_voltages_in.size(), {0.0f, 0.0f});

    BOOST_LOG_TRIVIAL(debug)
        << "Input TPA voltages to dedisperse, d_tpa_voltages_in.size(): "
        << d_tpa_voltages_in.size();
    BOOST_LOG_TRIVIAL(debug)
        << "Output FTPA voltages to write to, d_ftpa_voltages_out.size(): "
        << d_ftpa_voltages_out.size();

    thrust::transform(d_tpa_voltages_in.begin(),
                      d_tpa_voltages_in.end(),
                      _d_tpa_voltages_in_cufft.begin(),
                      [=] __device__(char2 const& val) {
                          hipfftComplex complex_val;
                          complex_val.x = val.x;
                          complex_val.y = val.y;
                          return complex_val;
                      });

    BOOST_LOG_TRIVIAL(debug) << "Transformed voltages to hipfftComplex";

    hipfftExecC2C(_config._fft_plan,
                 thrust::raw_pointer_cast(_d_tpa_voltages_in_cufft.data()),
                 thrust::raw_pointer_cast(_d_fpa_spectra.data()),
                 HIPFFT_FORWARD);

    BOOST_LOG_TRIVIAL(debug) << "Executed forward FFT";

    BOOST_LOG_TRIVIAL(debug) << "freq_idx = " << freq_idx;
    BOOST_LOG_TRIVIAL(debug) << "dm_idx = " << dm_idx;

    multiply_by_chirp(_d_fpa_spectra,
                      _d_fpa_spectra,
                      freq_idx,
                      dm_idx); // operating in place..

    BOOST_LOG_TRIVIAL(debug) << "Multiplied by chirp";

    hipfftExecC2C(_config._fft_plan,
                 thrust::raw_pointer_cast(_d_fpa_spectra.data()),
                 thrust::raw_pointer_cast(_d_tpa_voltages_dedispersed.data()),
                 HIPFFT_BACKWARD);

    BOOST_LOG_TRIVIAL(debug) << "Executed inverse FFT";

    std::size_t out_offset = freq_idx * _config.nantennas * _config.npols *
                             (_config.gulp_samps);
    std::size_t discard_size =
        _config.nantennas * _config.npols * _config.overlap_samps / 2;

    BOOST_LOG_TRIVIAL(debug) << "Output offset to write from: " << out_offset;
    BOOST_LOG_TRIVIAL(debug) << "discard_size: " << discard_size;
    BOOST_LOG_TRIVIAL(debug)
        << "copying from input from " << discard_size << " to "
        << _d_tpa_voltages_dedispersed.size() - discard_size;
    BOOST_LOG_TRIVIAL(debug)
        << "copying to output from " << out_offset << " to "
        << out_offset + _d_tpa_voltages_dedispersed.size() - 2 * discard_size;


    std::size_t fft_size  = _config.gulp_samps + _config.overlap_samps;


    // transform: divide by d_tpa_voltages_in.size()
    thrust::transform(_d_tpa_voltages_dedispersed.begin() + discard_size,
                      _d_tpa_voltages_dedispersed.end() - discard_size,
                      d_ftpa_voltages_out.begin() + out_offset,
                      [=] __device__(hipfftComplex const& val) {
                          char2 char2_val;
                          char2_val.x = static_cast<char>(
                              __float2int_rn(val.x / fft_size)); // scale the data back
                          char2_val.y =
                              static_cast<char>(__float2int_rn(val.y / fft_size));
                          return char2_val;
                      });
    d_ftpa_voltages_out.reference_dm(_config._h_dms[dm_idx]);
}

void CoherentDedisperser::multiply_by_chirp(

    thrust::device_vector<hipfftComplex> const& _d_fpa_spectra_in,
    thrust::device_vector<hipfftComplex>& _d_fpa_spectra_out,
    unsigned int freq_idx,
    unsigned int dm_idx)
{
    std::size_t total_chans     = _config._d_ism_responses[dm_idx].size();
    std::size_t response_offset = freq_idx * _config.gulp_samps;

    BOOST_LOG_TRIVIAL(debug) << "Freq idx: " << freq_idx;
    BOOST_LOG_TRIVIAL(debug) << "_config.gulp_samps: " << _config.gulp_samps;
    BOOST_LOG_TRIVIAL(debug) << "response_offset: " << response_offset;

    dim3 blockSize(_config.nantennas * _config.npols);
    dim3 gridSize(_config.gulp_samps / NCHANS_PER_BLOCK);
    kernels::dedisperse<<<gridSize, blockSize>>>(
        thrust::raw_pointer_cast(_config._d_ism_responses[dm_idx].data() +
                                 response_offset),
        thrust::raw_pointer_cast(_d_fpa_spectra_in.data()),
        thrust::raw_pointer_cast(_d_fpa_spectra_out.data()),
        total_chans);
}
} // namespace skyweaver
namespace skyweaver
{
namespace kernels
{

__global__ void dedisperse(hipfftComplex const* __restrict__ _d_ism_response,
                           hipfftComplex const* in,
                           hipfftComplex* out,
                           unsigned total_chans)
{
    const unsigned pa_size = blockDim.x;

    volatile __shared__ hipfftComplex response[NCHANS_PER_BLOCK];

    const unsigned block_start_chan_idx = blockIdx.x * NCHANS_PER_BLOCK;

    const unsigned remainder =
        min(total_chans - block_start_chan_idx, NCHANS_PER_BLOCK);

    for(int idx = threadIdx.x; idx < remainder; idx += pa_size) {
        hipfftComplex const temp = _d_ism_response[block_start_chan_idx + idx];
        response[idx].x         = temp.x;
        response[idx].y         = temp.y;
    }

    __syncthreads();

    /**
    Each block processes NANT * NPOL in parallel (one per thread)
    Each thread processes NCHANS_PER_BLOCK channels sequentially, for a given
    (iant, ipol)
    **/

    for(int block_ichan_idx = 0; block_ichan_idx < remainder;
        ++block_ichan_idx) {
        const int chan_idx = (block_start_chan_idx + block_ichan_idx) *
                                 pa_size    // get to the correct chan_idx
                             + threadIdx.x; // get to the correct (iant, ipol)
        out[chan_idx] = hipCmulf(response[block_ichan_idx], in[chan_idx]);
    }
}

struct DMResponse {
    int num_coarse_chans;
    double low_freq;
    double coarse_chan_bw;
    double fine_chan_bw;
    double dm_prefactor;
    int num_fine_chans;

    DMResponse(int num_coarse_chans,
               int num_fine_chans,
               double low_freq,
               double coarse_chan_bw,
               double fine_chan_bw,
               double dm_prefactor)
        : num_coarse_chans(num_coarse_chans), num_fine_chans(num_fine_chans),
          low_freq(low_freq), coarse_chan_bw(coarse_chan_bw),
          fine_chan_bw(fine_chan_bw), dm_prefactor(dm_prefactor)
    {
    }

    __device__ inline hipfftComplex operator()(int tid) const
    {
        int chan      = tid / num_fine_chans; // Coarse channel
        int fine_chan = tid % num_fine_chans; // fine channel

        double nu_0 = low_freq + chan * coarse_chan_bw -
                      0.5f * coarse_chan_bw; // + fine_chan * fine_chan_bw;

        double nu = fine_chan * fine_chan_bw; // fine_chan_freq

        double phase_prefactor = nu * nu * dm_prefactor;
        double phase =
            phase_prefactor / ((nu_0 + nu) * nu_0 * nu_0); // precalculate
        hipfftDoubleComplex weight;
        sincos(phase,
               &weight.y,
               &weight.x); // TO DO: test if it is not approximate
        hipfftComplex float_weight;
        float_weight.x = static_cast<float>(weight.x);
        float_weight.y = static_cast<float>(weight.y);

        return float_weight;
    }
};

} // namespace kernels

void get_dm_responses(CoherentDedisperserConfig& config,
                      double dm_prefactor,
                      thrust::device_vector<hipfftComplex>& response)
{
    BOOST_LOG_TRIVIAL(debug) << "Generating DM responses";
    thrust::device_vector<int> indices(config.num_coarse_chans *
                                       config.gulp_samps);
    thrust::sequence(indices.begin(), indices.end());

    // Apply the DMResponse functor using thrust's transform
    thrust::transform(indices.begin(),
                      indices.end(),
                      response.begin(),
                      kernels::DMResponse(config.num_coarse_chans,
                                          config.gulp_samps,
                                          config.low_freq,
                                          config.coarse_chan_bw,
                                          config.fine_chan_bw,
                                          dm_prefactor));
}

} // namespace skyweaver
